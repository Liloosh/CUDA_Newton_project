#include "hip/hip_runtime.h"
#include "NewtonSolverGPUFunctions.h"
#include "DataInitializer.h"

__global__ void NewtonSolverGPUFunctions::gpu_dummy_warmup() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 32) {
        double tmp = idx * 0.1;
    }
}

__global__ void NewtonSolverGPUFunctions::gpu_compute_func_values(double* points_d, double* indexes_d, double* vec_d, int MATRIX_SIZE, int version, int power) {
    int x_blocks_count = (MATRIX_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;
    int tidx = threadIdx.x;

    extern __shared__ double shared_points[];

    if (gidx < MATRIX_SIZE) {
        double value = 1.0;
        for (int i = 0; i < power; i++) {
            value *= points_d[gidx];
        }
        shared_points[threadIdx.x] = value * indexes_d[gidy * MATRIX_SIZE + gidx];
    }
    else {
        shared_points[threadIdx.x] = 0.0;
    }
    __syncthreads();


    if (BLOCK_SIZE >= 1024 && threadIdx.x < 512) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 512];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 512 && threadIdx.x < 256) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 256];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 256 && threadIdx.x < 128) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 128];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 128 && threadIdx.x < 64) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 64];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 64 && threadIdx.x < 32) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 32];
    }

    __syncthreads();

    if (threadIdx.x < 32) {

        if (version >= 7) {
            double sum = shared_points[threadIdx.x];
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 16);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 8);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 4);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 2);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 1);
            if (threadIdx.x == 0) {
                vec_d[gidy * x_blocks_count + blockIdx.x] = sum;
            }
        }
        else {
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 16]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 8]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 4]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 2]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 1]; __syncwarp();
            if (tidx == 0) {
                vec_d[gidy * x_blocks_count + blockIdx.x] = shared_points[threadIdx.x];
            }
        }
    }
}

__global__ void NewtonSolverGPUFunctions::gpu_compute_delta_values(double* points_d, double* indexes_d, double* vec_d, int MATRIX_SIZE, int version) {
    int x_blocks_count = (MATRIX_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;
    int tidx = threadIdx.x;

    extern __shared__ double shared_points[];

    if (gidx < MATRIX_SIZE) {
        shared_points[threadIdx.x] = points_d[gidx] * indexes_d[gidy * MATRIX_SIZE + gidx];
    }
    else {
        shared_points[threadIdx.x] = 0.0;
    }
    __syncthreads();


    if (BLOCK_SIZE >= 1024 && threadIdx.x < 512) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 512];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 512 && threadIdx.x < 256) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 256];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 256 && threadIdx.x < 128) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 128];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 128 && threadIdx.x < 64) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 64];
    }

    __syncthreads();

    if (BLOCK_SIZE >= 64 && threadIdx.x < 32) {
        shared_points[threadIdx.x] += shared_points[threadIdx.x + 32];
    }

    __syncthreads();

    if (threadIdx.x < 32) {

        if (version >= 7) {
            double sum = shared_points[threadIdx.x];
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 16);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 8);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 4);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 2);
            sum += __shfl_down_sync(SHAFFLE_CONST, sum, 1);
            if (threadIdx.x == 0) {
                vec_d[gidy * x_blocks_count + blockIdx.x] = sum;
            }
        }
        else {
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 16]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 8]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 4]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 2]; __syncwarp();
            shared_points[threadIdx.x] += shared_points[threadIdx.x + 1]; __syncwarp();
            if (tidx == 0) {
                vec_d[gidy * x_blocks_count + blockIdx.x] = shared_points[threadIdx.x];
            }
        }
    }
}

__global__ void NewtonSolverGPUFunctions::gpu_compute_jacobian(double* points_d, double* indexes_d, double* jacobian_d, int MATRIX_SIZE, int power) {
    extern __shared__ double shared_data[];

    int row = blockIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    double result = 0.0;
    double f_minus = 0.0;
    double f_plus = 0.0;

    for (int ph = 0; ph < gridDim.x; ++ph) {
        int global_col = ph * blockDim.x + threadIdx.x;

        if (global_col < MATRIX_SIZE) {
            shared_data[threadIdx.x] = points_d[global_col];
            shared_data[blockDim.x + threadIdx.x] = indexes_d[row * MATRIX_SIZE + global_col];
        }
        else {
            shared_data[threadIdx.x] = 0.0;
            shared_data[blockDim.x + threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (int i = 0; i < blockDim.x; ++i) {
            if (ph * blockDim.x + i >= MATRIX_SIZE) break;

            double value = shared_data[i];
            double element = shared_data[blockDim.x + i];

            if (ph * blockDim.x + i == col) {
                double x_value_plus = 1;
                double x_value_minus = 1;
                for (int i = 0; i < power; i++) {
                    x_value_plus *= (value + EQURENCY);
                    x_value_minus *= (value - EQURENCY);
                }
                f_minus += x_value_minus * element;
                f_plus += x_value_plus * element;
            }
            else {
                f_minus += value * element;
                f_plus += value * element;
            }
        }

        __syncthreads();
    }

    result = (f_plus - f_minus) / (2 * EQURENCY);

    if (row < MATRIX_SIZE && col < MATRIX_SIZE) {
        jacobian_d[row * MATRIX_SIZE + col] = result;
    }
}